#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "io.cuh"
#include "vector.cuh"
#include "random.cuh"
#include "normalmodel.cuh"
#include "regmodel.cuh"
#include "ar1model.cuh"
#include "dlm.cuh"
#include "sv.cuh"
#include "runfunc.cuh"
//
//
//
int main(int argc,char **argv)
{
  printf("Start .... \n");
  //
  //estimate_sv("sp500_ret_80_87.txt",-0.5,0.95,0.2);
  //estimate_svt("sp500_ret_80_87.txt",-0.5,0.95,0.2,20);
  //estimate_svl("sp500_ret_80_87.txt",-0.5,0.95,0.2,-0.5);
  //estimate_svtl("sp500_ret_80_87.txt",-0.5,0.95,0.2,-0.5,20);
  int flag = atoi(argv[1]);
  if(flag == 0) estimate_sv_gpu("sp500_ret_80_87.txt");
  if(flag == 1) estimate_svl_gpu("sp500_ret_80_87.txt");
  if(flag == 2) estimate_svtl_gpu("sp500_ret_80_87.txt");
  if(flag == 3) estimate_svt_gpu("sp500_ret_80_87.txt");
  //
  //
  printf("Done ... \n");
  //
  return 0;
}
