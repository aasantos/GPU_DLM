#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "io.cuh"
#include "vector.cuh"
#include "random.cuh"
#include "normalmodel.cuh"
#include "regmodel.cuh"
#include "ar1model.cuh"
#include "dlm.cuh"
#include "sv.cuh"
#include "runfunc.cuh"
//
//
//
int main()
{
  printf("Start .... \n");
  //
  estimate_sv("sp500_ret_80_87.txt",-0.5,0.95,0.2);
  estimate_svl("sp500_ret_80_87.txt",-0.5,0.95,0.2,-0.7);
  estimate_sv_gpu("sp500_ret_80_87.txt");
  //estimate_svl_gpu("sp500_ret_80_87.txt");
  printf("Done ... \n");
  //
  return 0;
}
